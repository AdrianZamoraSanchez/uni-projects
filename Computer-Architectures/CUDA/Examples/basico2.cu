/*
* ARQUITECTURA DE COMPUTADORES
* Hecho por: Adrián Zamora Sánchez
* EJEMPLO: "Básico 2 de CUDA"
* >> Generamos un array de N números aleatorios <0 >9 en host,
*    creamos otro array invirtiendo el anterior en dev, finalmente
*    sumamos ambos en dev y lo imprimimos desde host
*/

// INCLUDES
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <time.h>


// Función que muestra los datos del dispositivo
void propiedades_Device(int deviceID, int *maxThreads)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceID);

	// calculo del numero de cores (SP)
	int cudaCores = 0;
	int SM = deviceProp.multiProcessorCount;
	*maxThreads = deviceProp.maxThreadsPerBlock;
	int major = deviceProp.major;
	int minor = deviceProp.minor;
	const char* archName;

	switch (major)
	{
	case 1:
		//TESLA
		archName = "TESLA";
		cudaCores = 8;
		break;
	case 2:
		//FERMI
		archName = "FERMI";
		if (minor == 0)
			cudaCores = 32;
		else
			cudaCores = 48;
		break;
	case 3:
		//KEPLER
		archName = "KEPLER";
		cudaCores = 192;
		break;
	case 5:
		//MAXWELL
		archName = "MAXWELL";
		cudaCores = 128;
		break;
	case 6:
		//PASCAL
		archName = "PASCAL";
		cudaCores = 64;
		break;
	case 7:
		//VOLTA(7.0) //TURING(7.5)
		cudaCores = 64;
		if (minor == 0)
			archName = "VOLTA";
		else
			archName = "TURING";
		break;
	case 8:
		// AMPERE
		archName = "AMPERE";
		cudaCores = 64;
		break;
	case 9:
		//HOPPER
		archName = "HOPPER";
		cudaCores = 64;
		break;
	default:
		//ARQUITECTURA DESCONOCIDA
		archName = "DESCONOCIDA";
	}

	int rtV;
	hipRuntimeGetVersion(&rtV);

	// presentacion de propiedades
	printf("***************************************************\n");
	printf("DEVICE %d: %s\n", deviceID, deviceProp.name);
	printf("***************************************************\n");
	printf("> CUDA Toolkit\t\t\t: %d.%d\n", rtV / 1000, (rtV % 1000) / 10);
	printf("> Arquitectura CUDA\t\t: %s\n", archName);
	printf("> Capacidad de Computo\t\t: %d.%d\n", major, minor);
	printf("> No. MultiProcesadores\t\t: %d\n", SM);
	printf("> No. Nucleos CUDA (%dx%d)\t: %d\n", cudaCores, SM, cudaCores * SM);
	printf("> Memoria Global (total)\t: %u MiB\n", deviceProp.totalGlobalMem / (1024 * 1024));
	printf("***************************************************\n");
}

// Kernel del device donde se genera el vector inverso y se suman ambos
__global__ void sumarVectores(int* dev_vector1, int* dev_vector2, int* dev_vectorSuma,int N)
{
	int id = threadIdx.x;
	
	dev_vector2[id] = dev_vector1[N - 1 - id];
	
	dev_vectorSuma[id] = dev_vector2[id] + dev_vector1[id];
}

// Función donde el host genera el vector de tamaño N con valores aleatorios
__host__ void generarVector(int* hst_vector1, int N) { ;
	printf("Vector host: ");
	for (int i = 0; i < N; i++){
		hst_vector1[i] = rand() % 10;
		printf("%d ", hst_vector1[i]);
	}
}

// Función main
int main(int argc, char** argv)
{
	// Variable de control para no sobrepasar el maximo de hilos de la GPU
	int maxThreads;

	// Busca los dispositivos CUDA
	int deviceCount;
	
	hipGetDeviceCount(&deviceCount);

	// Comprueba que exista algún dispositivo
	if (deviceCount == 0)
	{
		// Sin no encuentra dispositivos compatibles con CUDA deuvelve un error
		printf("No se han encontrado dispositivos CUDA!\n");
		return 1;
	}
	else
	{
		// Muestra los datos del dispositivo
		for (int id = 0; id < deviceCount; id++)
		{
			propiedades_Device(id, &maxThreads);
		}
	}

	// Generamos aleatoriedad en rand()
	srand(static_cast<unsigned>(time(nullptr)));

	// Pedimos al usuario un número de elementos para los vectores
	int N;
	printf("Introduce el numero de elementos\n");
	std::cin >> N; // Tomamos el input del usuario

	// Comprobamos que el usuario no haya creado un vector con más elementos que hilos
	if (maxThreads < N) {
		std::cout << "Error, se ha sobrepasado el maximo de hilos: " << maxThreads;
		return 0;
	}

	// Definimos los vectores con los que trabaja el host
	int* hst_vector1, *hst_vector2, *hst_vectorSuma;

	// Reservamos la memoria de los vectores host
	hst_vector1 = (int*)malloc(N * sizeof(int));
	hst_vector2 = (int*)malloc(N * sizeof(int));
	hst_vectorSuma = (int*)malloc(N * sizeof(int));

	// Definimos los vectores con los que trabaja dev
	int* dev_vector1, * dev_vector2, *dev_vectorSuma;

	// Reservamos la memoria e los vectores dev
	hipMalloc(&dev_vector1, N * sizeof(int));
	hipMalloc(&dev_vector2, N * sizeof(int));
	hipMalloc(&dev_vectorSuma, N * sizeof(int));
	
	// Llamamos a una función __host__ que davalors alatorios al vector1
	generarVector(hst_vector1, N);

	// Pasamos el vector 1 al dev
	hipMemcpy(dev_vector1, hst_vector1, N * sizeof(int), hipMemcpyHostToDevice);

	// Invertimos el vector y hacemos la suma en dev
	sumarVectores <<<1,N>>> (dev_vector1, dev_vector2, dev_vectorSuma, N);

	// Copiamosl los resultados para su impresión por pantalla
	hipMemcpy(hst_vector2, dev_vector2, N * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_vectorSuma, dev_vectorSuma, N * sizeof(int), hipMemcpyDeviceToHost);

	// Imrpimimos el vector1 invertido (vector2)
	printf("\nVector device: ");
	for (int i = 0; i < N; i++) {
		printf("%d ", hst_vector2[i]);
	}

	// Imprimimos el valor de la suma (vectorSuma)
	printf("\nVector suma: ");
	for (int i = 0; i < N; i++) {
		printf("%d ", hst_vectorSuma[i]);
	}
	
	// Liberamos toda la memoria reservada en host y dev
	free(hst_vector1);
	free(hst_vector2);
	free(hst_vectorSuma);
	hipFree(dev_vector1);
	hipFree(dev_vector2);
	hipFree(dev_vectorSuma);

	// Hacemos un log que incluya la finalización de la ejecución del programa
	time_t fecha;
	time(&fecha);
	printf("\n***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));

	// Termina el programa
	return 0;
}
